// MIT License
// Copyright (c) 2020-2024 Sherman Lo

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// See EmpiricalNullFilter - this is the main kernel and the main point of entry
// Notes: row major
// Notes: the image to filter can be commonly referred to as the cache
// Notes: __constant__ are to be set before running the kernel
// Notes: shared memory is used to store the empirical null mean and std. IF big
//   enough, also the image. Size becomes a problem if the kernel radius becomes
//   too big, in this case, the image lives in global memory and hopefully may
//   be picked up in L1 and L2
__constant__ int kRoiWidth;      // region of interest width
__constant__ int kRoiHeight;     // region of interest height
__constant__ int kCacheWidth;    // width of the image (including padding)
__constant__ int kKernelRadius;  // the radius of the kernel
__constant__ int kKernelHeight;  // the number of rows in the kernel
__constant__ int kNInitial;      // number of initial values for Newton-Raphson
__constant__ int kNStep;         // number of steps for Newton-Raphson
__constant__ int kIsCopyImageToShared;  // indicate to copy image to shared mem

/**
 * Get derivative of the log density
 *
 * Set dx_lnf to contain derivatives of the density estimate evaluated at a
 * point
 *
 * @param cache_pointer the image to filter, can either be in global or shared
 *   memory, positioned at the centre of the kernel
 * @param cache_width the width of the image in cache_pointer
 * @param bandwidth parameter for the density estimate
 * @param kernel_pointers array (even number of elements, size 2*kKernelHeight)
 *   containing pairs of integers, indicates for each row the starting and
 *   ending column position from the centre of the kernel
 * @param value where the density estimate is evaluated at
 * @param dx_lnf MODIFIED 3-element array, to store results. The elements are:
 *   <ol>
 *     <li>the density (ignore any constant multiplied to it) (NOT THE LOG)</li>
 *     <li>the first derivative of the log density</li>
 *     <li>the second derivative of the log density</li>
 *   </ol>
 */
__device__ void GetDLnDensity(float* cache_pointer, int cache_width,
                              float bandwidth, int* kernel_pointers,
                              float* value, float* dx_lnf) {
  // variables when going through all pixels in the kernel
  float z;                       // value of a pixel when looping through kernel
  float sum_kernel[3] = {0.0f};  // store sums of weights
  float phi_z;                   // weight, use Gaussian kernel

  // pointer for the image
  // point to the top of the kernel
  cache_pointer -= kKernelRadius * cache_width;

  // for each row in the kernel
  for (int i = 0; i < 2 * kKernelHeight; i++) {
    // for each column for this row
    for (int dx = kernel_pointers[i++]; dx <= kernel_pointers[i]; dx++) {
      // append to sum if the value in cache_pointer is finite
      z = *(cache_pointer + dx);
      if (isfinite(z)) {
        z -= *value;
        z /= bandwidth;
        phi_z = expf(-z * z / 2);
        sum_kernel[0] += phi_z;
        sum_kernel[1] += phi_z * z;
        sum_kernel[2] += phi_z * z * z;
      }
    }
    cache_pointer += cache_width;
  }

  // work out derivatives
  float normaliser = bandwidth * sum_kernel[0];
  dx_lnf[0] = sum_kernel[0];
  dx_lnf[1] = sum_kernel[1] / normaliser;
  dx_lnf[2] = (sum_kernel[0] * (sum_kernel[2] - sum_kernel[0]) -
               sum_kernel[1] * sum_kernel[1]) /
              (normaliser * normaliser);
}

/**
 * Find mode
 *
 * Use Newton-Raphson to find the maximum value of the density estimate. Uses
 * the passed null_mean as the initial value and modifies it at each step,
 * ending up with a final answer.
 *
 * The second derivative of the log density and the density (up to a constant)
 * at the final answer is stored in second_diff_ln and density_at_mode.
 *
 * @param cache_pointer the image to filter, can either be in global or shared
 *   memory, positioned at the centre of the kernel
 * @param cache_width the width of the image in cache_pointer
 * @param bandwidth bandwidth for the density estimate
 * @param kernel_pointers array (even number of elements, size 2*kKernelHeight)
 *   containing pairs of integers, indicates for each row the starting and
 *   ending column position from the centre of the kernel
 * @param null_mean MODIFIED initial value for the Newton-Raphson method,
 *   modified to contain the final answer
 * @param second_diff_ln MODIFIED second derivative of the log density at the
 *   mode
 * @param density_at_mode MODIFIED contains the density (up to a constant) at
 *   the mode
 * @returns true if sucessful, false otherwise
 */
__device__ bool FindMode(float* cache_pointer, int cache_width, float bandwidth,
                         int* kernel_pointers, float* null_mean,
                         float* second_diff_ln, float* density_at_mode) {
  float dx_lnf[3];
  // kNStep of Newton-Raphson
  for (int i = 0; i < kNStep; i++) {
    GetDLnDensity(cache_pointer, cache_width, bandwidth, kernel_pointers,
                  null_mean, dx_lnf);
    *null_mean -= dx_lnf[1] / dx_lnf[2];
  }
  GetDLnDensity(cache_pointer, cache_width, bandwidth, kernel_pointers,
                null_mean, dx_lnf);
  // need to check if answer is valid
  if (isfinite(*null_mean) && isfinite(dx_lnf[0]) && isfinite(dx_lnf[1]) &&
      isfinite(dx_lnf[2]) && (dx_lnf[2] < 0)) {
    *density_at_mode = dx_lnf[0];
    *second_diff_ln = dx_lnf[2];
    return true;
  }
  return false;
}

/**
 * Copy image to shared memory
 *
 * @param dest pointer to shared memory
 * @param cache_in_block_width the size of the cache captured by the block,
 *   including the padding
 * @param source pointer to image
 * @param kernel_pointers array (even number of elements, size 2*kKernelHeight)
 *   containing pairs of integers, indicates for each row the starting and
 *   ending column position from the centre of the kernel
 */
__device__ void CopyImageToSharedMemory(float* dest, int cache_in_block_width,
                                        float* source, int* kernel_pointers) {
  // point to top left
  dest -= kKernelRadius * cache_in_block_width;
  source -= kKernelRadius * kCacheWidth;
  // for each row in the kernel
  for (int i = 0; i < 2 * kKernelHeight; i++) {
    // for each column for this row
    for (int dx = kernel_pointers[i++]; dx <= kernel_pointers[i]; dx++) {
      *(dest + dx) = *(source + dx);
    }
    source += kCacheWidth;
    dest += cache_in_block_width;
  }
}

/**
 * Main kernel: Empirical Null Filter
 *
 * Does the empirical null filter on the pixels in image, giving the empirical
 * null mean (aka mode) and the empirical null std.
 *
 * @param cache array of pixels to filter
 * @param initial_sigma_roi: array of pixels (same size as the ROI) containing
 *   standard deviations, used for producing random initial values for
 *   Newton-Raphson
 * @param bandwidth_roi array of pixels (same size as the ROI) containing the
 *   bandwidth for the density estimate
 * @param kernel_pointers: array (even number of elements, size 2*kKernelHeight)
 *   containing pairs of integers, indicates for each row the starting and
 *   ending column position from the centre of the kernel
 * @param null_mean_roi MODIFIED array of pixels (same size as ROI), pass
 *   results of median filter here to be used as initial values. Modified to
 *   contain the empricial null mean afterwards.
 * @param null_std_roi MODIFIED array of pixels (same size as ROI) to contain
 *   the empirical null std
 * @param progress_roi MODIFIED array of pixels (same size as ROI) initally
 *   contains all zeros. A filtered pixel will change it to a one.
 */
extern "C" __global__ void EmpiricalNullFilter(
    float* cache, float* initial_sigma_roi, float* bandwidth_roi,
    int* kernel_pointers, float* null_mean_roi, float* null_std_roi,
    int* progress_roi) {
  int x0 = threadIdx.x + blockIdx.x * blockDim.x;
  int y0 = threadIdx.y + blockIdx.y * blockDim.y;
  // adjust pointer to the corresponding x y coordinates
  cache += (y0 + kKernelRadius) * kCacheWidth + x0 + kKernelRadius;
  // check if in roi
  // &&isfinite(*cache) is not required as accessing the image from this
  // pixel is within bounds
  bool is_in_roi = x0 < kRoiWidth && y0 < kRoiHeight;

  // get shared memory
  extern __shared__ float shared_memory[];
  float* null_mean_shared_pointer = shared_memory;
  float* second_diff_shared_pointer =
      null_mean_shared_pointer + blockDim.x * blockDim.y;

  // offset by the x and y coordinates
  int roi_index = y0 * kRoiWidth + x0;
  int null_shared_index = threadIdx.y * blockDim.x + threadIdx.x;

  // cache_pointer points to the image to filter (including padding)
  // cache_pointer may either points to global or shared memory
  // cache_width will specify the width of the cache according to if the cache
  // is in global or shared memory
  float* cache_pointer;
  int cache_width;

  // if the shared memory is big enough, copy the image
  // cache_pointer points to shared memory if shared memory allows it, otherwise
  // points to global memory
  if (kIsCopyImageToShared) {
    // width of the cache captured by a block, including the padding
    // padding is of kKernelRadius size, on left and right
    cache_width = blockDim.x + 2 * kKernelRadius;
    cache_pointer = second_diff_shared_pointer + blockDim.x * blockDim.y;
    cache_pointer += (threadIdx.y + kKernelRadius) * cache_width + threadIdx.x +
                     kKernelRadius;
    // copy image to shared memory
    if (is_in_roi) {
      CopyImageToSharedMemory(cache_pointer, cache_width, cache,
                              kernel_pointers);
    }
  } else {
    // else keep the cache in global memory
    cache_width = kCacheWidth;
    cache_pointer = cache;
  }

  __syncthreads();

  // adjust pointer to the corresponding x y coordinates
  null_mean_shared_pointer += null_shared_index;
  second_diff_shared_pointer += null_shared_index;

  // for rng
  hiprandState_t state;
  hiprand_init(0, roi_index, 0, &state);
  // null_mean used to store mode for each initial value
  float null_mean;
  float median;
  float sigma;      // how much noise to add
  float bandwidth;  // bandwidth for density estimate

  if (is_in_roi) {
    null_mean = null_mean_roi[roi_index];  // use median as first initial
    median = null_mean;
    // modes with highest densities are stored in shared memory
    *null_mean_shared_pointer = null_mean;
    sigma = initial_sigma_roi[roi_index];  // how much noise to add
    bandwidth = bandwidth_roi[roi_index];  // bandwidth for density estimate
  }

  bool is_success;        // indicate if newton-raphson was sucessful
  float density_at_mode;  // density for this particular mode
  // second derivative of the log density, to set empirical null std
  float second_diff_ln;
  // keep solution with the highest density
  float max_density_at_mode = -INFINITY;

  // try different initial values, the first one is the median, then for
  // additional initial values, add normal noise to neighbouring null_mean
  // solutions in shared memory, neighbours rotate from -1, itself and +1 from
  // current pointer
  int min;
  int n_neighbour;
  float initial0;
  if (null_shared_index == 0) {
    min = 0;
  } else {
    min = -1;
  }
  if (null_shared_index == blockDim.x * blockDim.y - 1) {
    n_neighbour = 1 - min;
  } else {
    n_neighbour = 2 - min;
  }

  for (int i = 0; i < kNInitial; i++) {
    if (is_in_roi) {
      is_success =
          FindMode(cache_pointer, cache_width, bandwidth, kernel_pointers,
                   &null_mean, &second_diff_ln, &density_at_mode);
      // keep null_mean and nullStd with the highest density
      if (is_success) {
        if (density_at_mode > max_density_at_mode) {
          max_density_at_mode = density_at_mode;
          *null_mean_shared_pointer = null_mean;
          *second_diff_shared_pointer = second_diff_ln;
        }
      }
    }

    // try different initial value
    __syncthreads();

    if (is_in_roi) {
      // try an initial value using its neighbour in shared memory
      initial0 = *(null_mean_shared_pointer + i % n_neighbour + min);
      // ensure the initial value is finite, otherwise use previous solution
      if (!isfinite(initial0)) {
        initial0 = null_mean;
      }
      // add normal noise and add bias towards median
      null_mean = (initial0 + median) / 2 + sigma * hiprand_normal(&state);
    }
  }

  // store final results
  if (is_in_roi) {
    null_mean_roi[roi_index] = *null_mean_shared_pointer;
    null_std_roi[roi_index] = powf(-*second_diff_shared_pointer, -0.5f);
    progress_roi[roi_index] = 1;
  }
}

/**
 * Get the (local) count, mean and std in a kernel
 *
 * @param cache_pointer the image to filter on global memory, positioned at the
 *   centre of the kernel
 * @param kernel_pointers array (even number of elements, size 2*kKernelHeight)
 *   containing pairs of integers, indicates for each row the starting and
 *   ending column position from the centre of the kernel
 * @param count MODIFIED the resulting local count, ie number of finite elements
 *   in the kernel
 * @param mean MODIFIED the resulting local mean
 * @param std MODIFIED the resulting local std
 */
__device__ void GetMeanStd(float* cache_pointer, int* kernel_pointers,
                           int* count, float* mean, float* std) {
  float z;  // value of a pixel when looping through kernel

  // initial values
  *count = 0;
  *mean = {0.0f};
  *std = {0.0f};

  // pointer for the image
  // point to the top of the kernel
  float* cache_start = cache_pointer - kKernelRadius * kCacheWidth;

  cache_pointer = cache_start;

  // calculate count and mean here
  // for each row in the kernel
  for (int i = 0; i < 2 * kKernelHeight; i++) {
    // for each column for this row
    for (int dx = kernel_pointers[i++]; dx <= kernel_pointers[i]; dx++) {
      z = *(cache_pointer + dx);
      if (isfinite(z)) {
        ++(*count);
        *mean += z;
      }
    }
    cache_pointer += kCacheWidth;
  }
  *mean /= (float)*count;

  // given mean, calculate std
  cache_pointer = cache_start;
  // for each row in the kernel
  for (int i = 0; i < 2 * kKernelHeight; i++) {
    // for each column for this row
    for (int dx = kernel_pointers[i++]; dx <= kernel_pointers[i]; dx++) {
      z = *(cache_pointer + dx);
      if (isfinite(z)) {
        *std += (z - *mean) * (z - *mean);
      }
    }
    cache_pointer += kCacheWidth;
  }
  *std /= (float)(*count - 1);
  *std = sqrtf(*std);
}

/**
 * Kernel: Mean and Standard Deviation Filter
 *
 * Does the mean and standard deviation filter on an image. It ignore non-finite
 * elements. Also returns the local number of finite elements in the kernel.
 * Non-finite elements occur at the padding.
 *
 * @param cache array of pixels to filter
 * @param kernel_pointers: array (even number of elements, size 2*kKernelHeight)
 *   containing pairs of integers, indicates for each row the starting and
 *   ending column position from the centre of the kernel
 * @param count_roi MODIFIED array of pixels (same size as ROI), pass results
 *   with the local number of finite elements
 * @param mean_roi MODIFIED array of pixels (same size as ROI), pass results of
 *   the mean filter
 * @param std_roi MODIFIED array of pixels (same size as ROI), pass results of
 *   the std filter
 */
extern "C" __global__ void MeanStdFilter(float* cache, int* kernel_pointers,
                                         int* count_roi, float* mean_roi,
                                         float* std_roi) {
  int x0 = threadIdx.x + blockIdx.x * blockDim.x;
  int y0 = threadIdx.y + blockIdx.y * blockDim.y;
  // adjust pointer to the corresponding x y coordinates
  cache += (y0 + kKernelRadius) * kCacheWidth + x0 + kKernelRadius;
  // check if in roi
  bool is_in_roi = x0 < kRoiWidth && y0 < kRoiHeight;

  // offset by the x and y coordinates
  int roi_index = y0 * kRoiWidth + x0;

  if (is_in_roi) {
    GetMeanStd(cache, kernel_pointers, count_roi + roi_index,
               mean_roi + roi_index, std_roi + roi_index);
  }
}
